#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matrixMultiply(float *a, float *b, float *result, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;

    for (int i = 0; i < n; i++) {
        sum += a[row * n + i] * b[i * n + col];
    }
    result[row * n + col] = sum;
}

int main() {
    int n = 16;
    float a[n * n], b[n * n], result[n * n];
    // Initialize 'a' and 'b' with values here

    float *d_a, *d_b, *d_result;
    hipMalloc(&d_a, n * n * sizeof(float));
    hipMalloc(&d_b, n * n * sizeof(float));
    hipMalloc(&d_result, n * n * sizeof(float));

    hipMemcpy(d_a, a, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * n * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(n, n);
    matrixMultiply<<<1, threadsPerBlock>>>(d_a, d_b, d_result, n);

    hipMemcpy(result, d_result, n * n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    std::cout << "Matrix Multiplication Result:" << std::endl;
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            std::cout << result[i * n + j] << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}
