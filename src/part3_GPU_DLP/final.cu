#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matrixMultiplyOptimized(float *a, float *b, float *result, int n) {
    __shared__ float sharedA[16][16];
    __shared__ float sharedB[16][16];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;

    for (int i = 0; i < n / 16; i++) {
        sharedA[threadIdx.y][threadIdx.x] = a[row * n + (i * 16 + threadIdx.x)];
        sharedB[threadIdx.y][threadIdx.x] = b[(i * 16 + threadIdx.y) * n + col];
        __syncthreads();

        for (int k = 0; k < 16; k++) {
            sum += sharedA[threadIdx.y][k] * sharedB[k][threadIdx.x];
        }
        __syncthreads();
    }

    result[row * n + col] = sum;
}

int main() {
    int n = 1024;
    float *a, *b, *result;
    size_t size = n * n * sizeof(float);

    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&result, size);

    for (int i = 0; i < n * n; ++i) {
        a[i] = 1.0f; 
        b[i] = 2.0f;
    }

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(n / threadsPerBlock.x, n / threadsPerBlock.y);

    matrixMultiplyOptimized<<<blocksPerGrid, threadsPerBlock>>>(a, b, result, n);
    hipDeviceSynchronize();

    hipFree(a);
    hipFree(b);
    hipFree(result);

    return 0;
}
