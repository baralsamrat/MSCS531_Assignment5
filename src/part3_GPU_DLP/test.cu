#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void matrixMultiply(float *a, float *b, float *result, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;

    for (int i = 0; i < n; i++) {
        sum += a[row * n + i] * b[i * n + col];
    }
    result[row * n + col] = sum;
}

int main() {
    int n = 1024;  // Large matrix size
    float *a, *b, *result;
    size_t size = n * n * sizeof(float);

    // Allocate and initialize matrices
    hipMallocManaged(&a, size);
    hipMallocManaged(&b, size);
    hipMallocManaged(&result, size);
    for (int i = 0; i < n * n; ++i) {
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Configure CUDA grid and block dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(n / threadsPerBlock.x, n / threadsPerBlock.y);

    auto start = std::chrono::high_resolution_clock::now();
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(a, b, result, n);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float> duration = end - start;

    std::cout << "Matrix Multiplication Execution Time: " << duration.count() << " seconds\n";

    hipFree(a);
    hipFree(b);
    hipFree(result);

    return 0;
}
